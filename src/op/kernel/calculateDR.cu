#include <iostream>
#include <ATen/cuda/HIPContext.h>

#include <calculate_DR.h>

template <>
void launch_calculate_DR(
    float *xyz_scater, // batchsize x natoms x 4 x embedingnet_output_dim
    const int batch_size,
    const int natoms,
    const int neigh_num,
    const int ntype,
    const int embedingnet_output_dim,
    float *DR // batchsize x natoms x embedingnet_output_dim x 16
)
{
    hipblasStatus_t stat;

    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();

    float scale = float(1.0) / float(neigh_num * ntype);

    // stat = hipblasSscal(handle, batch_size * natoms * 4 * embedingnet_output_dim, &scale, xyz_scater, 1);

    // if (stat != HIPBLAS_STATUS_SUCCESS)
    // {
    //     std::cout << "CUBLAS scale failed\n";
    //     return;
    // }
    // float alpha = 1.0, beta = 0.0;

    int batchCount = batch_size * natoms;
    float alpha = scale * scale, beta = 0.0;

    // batchCount x 16 x 4, batchCount x 4 x 25
    stat = hipblasSgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 16, embedingnet_output_dim, 4, &alpha, xyz_scater, embedingnet_output_dim, 4 * embedingnet_output_dim,
                                     xyz_scater, embedingnet_output_dim, 4 * embedingnet_output_dim,
                                     &beta, DR, embedingnet_output_dim, embedingnet_output_dim * 16, batchCount);

    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS gemm failed\n";
        return;
    }

    return;
}

template <>
void launch_calculate_DR(
    double *xyz_scater, // batchsize x natoms x 4 x embedingnet_output_dim
    const int batch_size,
    const int natoms,
    const int neigh_num,
    const int ntype,
    const int embedingnet_output_dim,
    double *DR // batchsize x natoms x embedingnet_output_dim x 16
)
{
    hipblasStatus_t stat;
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();

    double scale = double(1.0) / double(neigh_num * ntype);

    // stat = hipblasDscal(handle, batch_size * natoms * 4 * embedingnet_output_dim, &scale, xyz_scater, 1);

    // if (stat != HIPBLAS_STATUS_SUCCESS)
    // {
    //     std::cout << "CUBLAS scale failed\n";
    //     return;
    // }

    // double alpha = 1.0, beta = 0.0;

    int batchCount = batch_size * natoms;

    double alpha = scale * scale, beta = 0.0;

    // batchCount x 16 x 4, batchCount x 4 x 25
    stat = hipblasDgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                                     16, embedingnet_output_dim, 4,
                                     &alpha, xyz_scater, embedingnet_output_dim, 4 * embedingnet_output_dim,
                                     xyz_scater, embedingnet_output_dim, 4 * embedingnet_output_dim,
                                     &beta, DR, 16, embedingnet_output_dim * 16, batchCount);

    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS gemm failed\n";
        return;
    }

    return;
}

template <>
void launch_calculate_DR_grad(
    const float *xyz_scater,
    const int batch_size,
    const int natoms,
    const int neigh_num,
    const int ntype,
    const int embedingnet_output_dim,
    const float *grad_output,
    float *grad)
{
    hipblasStatus_t stat;
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();

    float scale = float(1.0) / float(neigh_num * ntype);

    float alpha = scale * scale, beta = 0.0;

    int batchCount = batch_size * natoms;

    // batchCount x 4 x 16, batchCount x 25 x 16
    stat = hipblasSgemmStridedBatched(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                                     embedingnet_output_dim, 4, 16,
                                     &alpha, grad_output, 16, 16 * embedingnet_output_dim,
                                     xyz_scater, embedingnet_output_dim, 4 * embedingnet_output_dim,
                                     &beta, grad, embedingnet_output_dim, embedingnet_output_dim * 4, batchCount);

    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS gemm failed\n";
        return;
    }

    beta = 1.0;
    stat = hipblasSgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                                     16, 4, embedingnet_output_dim,
                                     &alpha, grad_output, 16, 16 * embedingnet_output_dim,
                                     xyz_scater, embedingnet_output_dim, 4 * embedingnet_output_dim,
                                     &beta, grad, embedingnet_output_dim, embedingnet_output_dim * 4, batchCount);

    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS gemm failed\n";
        return;
    }
}

template <>
void launch_calculate_DR_grad(
    const double *xyz_scater,
    const int batch_size,
    const int natoms,
    const int neigh_num,
    const int ntype,
    const int embedingnet_output_dim,
    const double *grad_output,
    double *grad)
{
    hipblasStatus_t stat;
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();

    double scale = double(1.0) / double(neigh_num * ntype);

    scale = scale * scale;

    // stat = hipblasDscal(handle, batch_size * natoms * 4 * embedingnet_output_dim, &scale, xyz_scater, 1);

    // if (stat != HIPBLAS_STATUS_SUCCESS)
    // {
    //     std::cout << "CUBLAS scale failed\n";
    //     return;
    // }

    // double alpha = 1.0, beta = 0.0;
    double alpha = scale, beta = 0.0;

    int batchCount = batch_size * natoms;

    // batchCount x 25 x 16   batchCount x 16 x 4
    stat = hipblasDgemmStridedBatched(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                                     embedingnet_output_dim, 4, 16,
                                     &alpha, grad_output, 16, 16 * embedingnet_output_dim,
                                     xyz_scater, embedingnet_output_dim, 4 * embedingnet_output_dim,
                                     &beta, grad, embedingnet_output_dim, embedingnet_output_dim * 4, batchCount);

    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS gemm failed\n";
        return;
    }

    beta = 1.0;
    // batchCount x 16 x 25   batchCount x 25 x 4
    stat = hipblasDgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                     16, 4, embedingnet_output_dim,
                                     &alpha, grad_output, 16, 16 * embedingnet_output_dim,
                                     xyz_scater, embedingnet_output_dim, 4 * embedingnet_output_dim,
                                     &beta, grad, embedingnet_output_dim, embedingnet_output_dim * 4, batchCount);

    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS gemm failed\n";
        return;
    }
}

// A, B, C is Rowmajor
hipblasStatus_t cublasDgemmStrideBatchedRowMajor(hipblasHandle_t &handle, hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k,
                                                const double *alpha,
                                                const double *A, int lda,
                                                long long int strideA,
                                                const double *B, int ldb,
                                                long long int strideB,
                                                const double *beta,
                                                double *C, int ldc,
                                                long long int strideC,
                                                int batchCount)
{
    return hipblasDgemmStridedBatched(handle, transb, transa, n, m, k, alpha, B, ldb, strideB, A, lda, strideA, beta, C, ldc, strideC, batchCount);
}

hipblasStatus_t cublasSgemmStrideBatchedRowMajor(hipblasHandle_t &handle, hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k,
                                                const float *alpha,
                                                const float *A, int lda,
                                                long long int strideA,
                                                const float *B, int ldb,
                                                long long int strideB,
                                                const float *beta,
                                                float *C, int ldc,
                                                long long int strideC,
                                                int batchCount)
{
    return hipblasSgemmStridedBatched(handle, transb, transa, n, m, k, alpha, B, ldb, strideB, A, lda, strideA, beta, C, ldc, strideC, batchCount);
}

template <>
void launch_calculate_DR_second_grad(
    const int batch_size,
    const int natoms,
    const float scale,
    const int embedingnet_output_dim,
    const float *xyz_scater,
    const float *grad_output,
    const float *grad_second,
    float *dgrad_xyz_scater,
    float *dgrad_gradoutput)
{
    hipblasStatus_t stat;
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();

    int batchCount = batch_size * natoms;
    float alpha = scale, beta = 0.0;

    stat = cublasSgemmStrideBatchedRowMajor(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 4, embedingnet_output_dim, 16,
                                            &alpha, grad_second, embedingnet_output_dim, embedingnet_output_dim * 4,
                                            grad_output, 16, 16 * embedingnet_output_dim,
                                            &beta, dgrad_xyz_scater, embedingnet_output_dim, embedingnet_output_dim * 4, batchCount);

    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS gemm failed\n";
        return;
    }

    stat = cublasSgemmStrideBatchedRowMajor(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, embedingnet_output_dim, 16, 4,
                                            &alpha, grad_second, embedingnet_output_dim, embedingnet_output_dim * 4,
                                            xyz_scater, embedingnet_output_dim, 4 * embedingnet_output_dim,
                                            &beta, dgrad_gradoutput, 16, embedingnet_output_dim * 16, batchCount);
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS gemm failed\n";
        return;
    }

    beta = 1.0;

    stat = cublasSgemmStrideBatchedRowMajor(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, 16, embedingnet_output_dim,
                                            &alpha, grad_second, embedingnet_output_dim, embedingnet_output_dim * 4,
                                            grad_output, 16, 16 * embedingnet_output_dim,
                                            &beta, dgrad_xyz_scater, embedingnet_output_dim, embedingnet_output_dim * 4, batchCount);

    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS gemm failed\n";
        return;
    }

    stat = cublasSgemmStrideBatchedRowMajor(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, embedingnet_output_dim, 16, 4,
                                            &alpha, xyz_scater, embedingnet_output_dim, embedingnet_output_dim * 4,
                                            grad_second, embedingnet_output_dim, 4 * embedingnet_output_dim,
                                            &beta, dgrad_gradoutput, 16, embedingnet_output_dim * 16, batchCount);
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS gemm failed\n";
        return;
    }

    return;
}

template <>
void launch_calculate_DR_second_grad(
    const int batch_size,
    const int natoms,
    const double scale,
    const int embedingnet_output_dim,
    const double *xyz_scater,
    const double *grad_output,
    const double *grad_second,
    double *dgrad_xyz_scater,
    double *dgrad_gradoutput)
{
    hipblasStatus_t stat;
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();

    int batchCount = batch_size * natoms;
    double alpha = scale, beta = 0.0;

    stat = cublasDgemmStrideBatchedRowMajor(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 4, embedingnet_output_dim, 16,
                                            &alpha, grad_second, embedingnet_output_dim, embedingnet_output_dim * 4,
                                            grad_output, 16, 16 * embedingnet_output_dim,
                                            &beta, dgrad_xyz_scater, embedingnet_output_dim, embedingnet_output_dim * 4, batchCount);

    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS gemm failed\n";
        return;
    }

    stat = cublasDgemmStrideBatchedRowMajor(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, embedingnet_output_dim, 16, 4,
                                            &alpha, grad_second, embedingnet_output_dim, embedingnet_output_dim * 4,
                                            xyz_scater, embedingnet_output_dim, 4 * embedingnet_output_dim,
                                            &beta, dgrad_gradoutput, 16, embedingnet_output_dim * 16, batchCount);
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS gemm failed\n";
        return;
    }

    beta = 1.0;

    stat = cublasDgemmStrideBatchedRowMajor(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, 16, embedingnet_output_dim,
                                            &alpha, grad_second, embedingnet_output_dim, embedingnet_output_dim * 4,
                                            grad_output, 16, 16 * embedingnet_output_dim,
                                            &beta, dgrad_xyz_scater, embedingnet_output_dim, embedingnet_output_dim * 4, batchCount);

    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS gemm failed\n";
        return;
    }

    stat = cublasDgemmStrideBatchedRowMajor(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, embedingnet_output_dim, 16, 4,
                                            &alpha, xyz_scater, embedingnet_output_dim, embedingnet_output_dim * 4,
                                            grad_second, embedingnet_output_dim, 4 * embedingnet_output_dim,
                                            &beta, dgrad_gradoutput, 16, embedingnet_output_dim * 16, batchCount);
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS gemm failed\n";
        return;
    }

    return;
}